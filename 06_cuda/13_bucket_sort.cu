#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// #define DEBUG 1

__global__ void scan(int *a, int *b, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  grid_group grid = this_grid();
  for(int j=1; j<N; j<<=1) {
    b[i] = a[i];
    grid.sync();
    a[i] += b[i-j];
    grid.sync();
  }
}

__global__ void init_bucket(int *bucket, int range) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i >= range) return;
  bucket[i] = 0;
}

__global__ void count_key(int *key, int *bucket, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i >= n) return;
  atomicAdd(bucket + key[i], 1);
}

__global__ void write_key(int *key, int start, int end, int val, int n) {
  // bucket[start:end] = val;
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i >= n) return;
  if (start <= i && i < end) key[i] = val;
}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket, *buf;
  hipMallocManaged(&bucket, range*sizeof(int));  
  hipMallocManaged(&buf, range*sizeof(int));  

  int m = 1024;
  init_bucket<<<(range + m - 1)/m, m>>>(bucket, range);
  hipDeviceSynchronize();

  count_key<<<(n + m - 1)/m, m>>>(key, bucket, n);
  hipDeviceSynchronize();

#ifdef DEBUG
  // bucket before scan
  for (int i=0; i<range; i++) {
    printf("%d ", bucket[i]);
  }
  printf("\n");
#endif

  // bucket を scan する
  void *args[] = {(void *)&bucket, (void *)&buf, (void *)&n};
  hipLaunchCooperativeKernel((void*)scan, (n + m - 1)/m, m, args);
  hipDeviceSynchronize();

#ifdef DEBUG
  // bucket after scan
  for (int i=0; i<range; i++) {
    printf("%d ", bucket[i]);
  }
  printf("\n");
#endif

  for (int val=0; val<range; val++) {
    if (val == 0) {
      write_key<<<(n + m - 1)/m, m>>>(key, 0, bucket[val], val, n);
    } else {
      write_key<<<(n + m - 1)/m, m>>>(key, bucket[val-1], bucket[val], val, n);
    }

  }
  // それぞれ異なる場所に書き込むため、毎回同期を取る必要はない
  // write_key() 内で同期を取る必要もない
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(bucket);
  hipFree(key);
}
